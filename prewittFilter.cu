#include "hip/hip_runtime.h"
#include <thread>
#include <chrono>
#include <time.h>
#include <iostream>
#include <math.h>
#include "imageLoader.cpp"

#define GRIDVAL 20.0 

void prewitt_cpu(const byte* orig, byte* cpu, const unsigned int width, const unsigned int height);

/************************************************************************************************
 * void prewitt_gpu(const byte*, byte*, uint, uint);
 * - This function runs on the GPU, it works on a 2D grid giving the current x, y pair being worked
 * - on, the const byte* is the original image being processed and the second byte* is the image
 * - being created using the prewitt filter. This function runs through a given x, y pair and uses 
 * - a prewitt filter to find whether or not the current pixel is an edge, the more of an edge it is
 * - the higher the value returned will be
 * 
 * Inputs: const byte* orig : the original image being evaluated
 *                byte* cpu : the image being created using the prewitt filter
 *               uint width : the width of the image
 *              uint height : the height of the image
 * 
 ***********************************************************************************************/
__global__ void prewitt_gpu(const byte* orig, byte* cpu, const unsigned int width, const unsigned int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    float dx, dy;
    if( x > 0 && y > 0 && x < width-1 && y < height-1) {
        dx = (-1* orig[(y-1)*width + (x-1)]) + (-2*orig[y*width+(x-1)]) + (-1*orig[(y+1)*width+(x-1)]) +
             (    orig[(y-1)*width + (x+1)]) + ( 2*orig[y*width+(x+1)]) + (   orig[(y+1)*width+(x+1)]);
        dy = (    orig[(y-1)*width + (x-1)]) + ( 2*orig[(y-1)*width+x]) + (   orig[(y-1)*width+(x+1)]) +
             (-1* orig[(y+1)*width + (x-1)]) + (-2*orig[(y+1)*width+x]) + (-1*orig[(y+1)*width+(x+1)]);
        cpu[y*width + x] = sqrt( (dx*dx) + (dy*dy) );
    }
}

/************************************************************************************************
 * int main(int, char*[])
 * - This function is our program's entry point. The function passes in the command line arguments
 * - and if there are exactly 2 command line arguments, the program will continue, otherwise it
 * - will exit with error code 1. If the program continues, it will read in the file given by 
 * - command line argument #2 and store as an array of bytes, after some header information is
 * - outputted, the prewitt filter will run in 3 different functions on the original image and
 * - 3 new images will be created, each containing a prewitt filter created using just the CPU, 
 * - OMP, and the GPU, then the image will be written out to a file with an appropriate indicator
 * - appended to the end of the filename.
 * 
 * Inputs:    int argc : the number of command line arguments
 *         char*argv[] : an array containing the command line arguments
 * Outputs:   returns 0: code ran successful, no issues came up
 *            returns 1: invalid number of command line arguments
 *            returns 2: unable to process input image
 *            returns 3: unable to write output image
 * 
 ***********************************************************************************************/
int main(int argc, char*argv[]) {
    /** Check command line arguments **/
    if(argc != 2) {
        printf("%s: Invalid number of command line arguments. Exiting program\n", argv[0]);
        printf("Usage: %s [image.png]", argv[0]);
        return 1;
    }
    /** Gather CUDA device properties **/
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	int cores = devProp.multiProcessorCount;
	switch (devProp.major)
	{
	case 2: // Fermi
		if (devProp.minor == 1) cores *= 48;
		else cores *= 32; break;
	case 3: // Kepler
		cores *= 192; break;
	case 5: // Maxwell
		cores *= 128; break;
	case 6: // Pascal
		if (devProp.minor == 1) cores *= 128;
		else if (devProp.minor == 0) cores *= 64;
		break;
    }
    
    /** Print out some header information (# of hardware threads, GPU info, etc) **/
    time_t rawTime;time(&rawTime);
    struct tm* curTime = localtime(&rawTime);
    char timeBuffer[80] = "";
    strftime(timeBuffer, 80, "edge map benchmarks (%c)\n", curTime);
    printf("%s", timeBuffer);
    printf("CPU: %d hardware threads\n", std::thread::hardware_concurrency());
    printf("GPGPU: %s, CUDA %d.%d, %zd Mbytes global memory, %d CUDA cores\n",
    devProp.name, devProp.major, devProp.minor, devProp.totalGlobalMem / 1048576, cores);

    /** Load our img and allocate space for our modified images **/
    imgData origImg = loadImage(argv[1]);
    imgData cpuImg(new byte[origImg.width*origImg.height], origImg.width, origImg.height);
  
    imgData gpuImg(new byte[origImg.width*origImg.height], origImg.width, origImg.height);
    
    /** make sure all our newly allocated data is set to 0 **/
    memset(cpuImg.pixels, 0, (origImg.width*origImg.height));
  

    /** We first run the prewitt filter on just the CPU using only 1 thread **/
    auto c = std::chrono::system_clock::now();
    prewitt_cpu(origImg.pixels, cpuImg.pixels, origImg.width, origImg.height);
    std::chrono::duration<double> time_cpu = std::chrono::system_clock::now() - c;

    /** Next, we use OpenMP to parallelize it **/
    c = std::chrono::system_clock::now();
  
 

    /** Finally, we use the GPU to parallelize it further **/
    /** Allocate space in the GPU for our original img, new img, and dimensions **/
    byte *gpu_orig, *gpu_prewitt;
    hipMalloc( (void**)&gpu_orig, (origImg.width * origImg.height));
    hipMalloc( (void**)&gpu_prewitt, (origImg.width * origImg.height));
    /** Transfer over the memory from host to device and memset the prewitt array to 0s **/
    hipMemcpy(gpu_orig, origImg.pixels, (origImg.width*origImg.height), hipMemcpyHostToDevice);
    hipMemset(gpu_prewitt, 0, (origImg.width*origImg.height));
   
    /** set up the dim3's for the gpu to use as arguments (threads per block & num of blocks)**/
    dim3 threadsPerBlock(GRIDVAL, GRIDVAL, 1);
    dim3 numBlocks(ceil(origImg.width/GRIDVAL), ceil(origImg.height/GRIDVAL), 1);

    /** Run the prewitt filter using the CPU **/
    c = std::chrono::system_clock::now();
    prewitt_gpu<<<numBlocks, threadsPerBlock>>>(gpu_orig, gpu_prewitt, origImg.width, origImg.height);
    hipError_t cudaerror = hipDeviceSynchronize(); // waits for completion, returns error code
    if ( cudaerror != hipSuccess ) fprintf( stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName( cudaerror ) ); // if error, output error
    std::chrono::duration<double> time_gpu = std::chrono::system_clock::now() - c;
    /** Copy data back to CPU from GPU **/
    hipMemcpy(gpuImg.pixels, gpu_prewitt, (origImg.width*origImg.height), hipMemcpyDeviceToHost);

    /** Output runtimes of each method of prewitt filtering **/
    printf("\nProcessing %s: %d rows x %d columns\n", argv[1], origImg.height, origImg.width);
    printf("CPU execution time    = %*.1f msec\n", 5, 1000*time_cpu.count());

    printf("CUDA execution time   = %*.1f msec\n", 5, 1000*time_gpu.count());

    printf("\nCPU->GPU speedup:%*.1f X", 12, (1000*time_cpu.count())/(1000*time_gpu.count()));
    printf("\n");

    /** Output the images of each prewitt filter with an appropriate string appended to the original image name **/
    writeImage(argv[1], "gpu", gpuImg);
    writeImage(argv[1], "cpu", cpuImg);


    /** Free any memory leftover.. gpuImig, cpuImg get their pixels free'd while writing **/
    hipFree(gpu_orig); hipFree(gpu_prewitt);
    return 0;
}

/************************************************************************************************
 * void prewitt_cpu(const byte*, byte*, uint, uint);
 * - This function runs on just the CPU with nothing running in parallel. The function takes in 
 * - an original image and compares the pixels to the left and right and then above and below
 * - to find the rate of change of the two comparisons, then squares, adds, and square roots the
 * - pair to find a 'prewitt' value, this value is saved into an array of bytes and then loops to
 * - handle the next pixel. The resulting array of evaluated pixels should be of an image showing
 * - in black and white where edges appear in the original image.
 * 
 * Inputs: const byte* orig : the original image being evaluated
 *                byte* cpu : the image being created using the prewitt filter
 *               uint width : the width of the image
 *              uint height : the height of the image
 * 
 ***********************************************************************************************/
void prewitt_cpu(const byte* orig, byte* cpu, const unsigned int width, const unsigned int height) {
    for(int y = 1; y < height-1; y++) {
        for(int x = 1; x < width-1; x++) {
            int dx = (-1*orig[(y-1)*width + (x-1)]) + (-2*orig[y*width+(x-1)]) + (-1*orig[(y+1)*width+(x-1)]) +
                 (orig[(y-1)*width + (x+1)]) + (2*orig[y*width+(x+1)]) + (orig[(y+1)*width+(x+1)]);
            int dy = (orig[(y-1)*width + (x-1)]) + (2*orig[(y-1)*width+x]) + (orig[(y-1)*width+(x+1)]) +
            (-1*orig[(y+1)*width + (x-1)]) + (-2*orig[(y+1)*width+x]) + (-1*orig[(y+1)*width+(x+1)]);
            cpu[y*width + x] = sqrt((dx*dx)+(dy*dy));
        }
    }
}


